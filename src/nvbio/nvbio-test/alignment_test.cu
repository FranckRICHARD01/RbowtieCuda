#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// alignment_test.cu
//

#define NVBIO_CUDA_DEBUG
#define NVBIO_CUDA_ASSERTS
//#define NVBIO_CUDA_NON_BLOCKING_ASSERTS
#define WFA_TESTS

#include <nvbio/alignment/wfa.h>
#include <nvbio-test/alignment_test_utils.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/cuda/ldg.h>
#include <nvbio/basic/cached_iterator.h>
#include <nvbio/basic/packedstream.h>
#include <nvbio/basic/packedstream_loader.h>
#include <nvbio/basic/vector_view.h>
#include <nvbio/basic/vector.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/basic/dna.h>
#include <nvbio/alignment/alignment.h>
#include <nvbio/alignment/batched.h>
#include <nvbio/alignment/sink.h>
#include <thrust/device_vector.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>

using namespace nvbio;

namespace nvbio
{
    namespace aln
    {

        enum
        {
            CACHE_SIZE = 32
        };
        typedef nvbio::lmem_cache_tag<CACHE_SIZE> lmem_cache_tag_type;
        typedef nvbio::uncached_tag uncached_tag_type;

        //
        // An alignment stream class to be used in conjunction with the BatchAlignmentScore class
        //
        template <typename t_aligner_type, uint32 M, uint32 N, typename cache_type = lmem_cache_tag_type>
        struct AlignmentStream
        {
            typedef t_aligner_type aligner_type;

            typedef nvbio::nvbio_cuda::ldg_pointer<uint32> storage_iterator;

            typedef nvbio::PackedStringLoader<storage_iterator, 4, false, cache_type> pattern_loader_type;
            typedef typename pattern_loader_type::input_iterator uncached_pattern_iterator;
            typedef typename pattern_loader_type::iterator pattern_iterator;
            typedef nvbio::vector_view<pattern_iterator> pattern_string;

            typedef nvbio::PackedStringLoader<storage_iterator, 2, false, cache_type> text_loader_type;
            typedef typename text_loader_type::input_iterator uncached_text_iterator;
            typedef typename text_loader_type::iterator text_iterator;
            typedef nvbio::vector_view<text_iterator> text_string;

            // typedef aln::wfa_type<int32>                                                      wfa_type;

            // an alignment context
            struct context_type
            {
                int32 min_score;
                aln::BestSink<int32> sink;
            };
            // a container for the strings to be aligned
            struct strings_type
            {
                pattern_loader_type pattern_loader;
                text_loader_type text_loader;
                pattern_string pattern;
                trivial_quality_string quals;
                text_string text;
            };

            // constructor
            AlignmentStream(
                aligner_type _aligner,
                const uint32 _count,
                const uint32 *_patterns,
                const uint32 *_text,
                int16 *_scores) : m_aligner(_aligner), m_count(_count), m_patterns(storage_iterator(_patterns)), m_text(storage_iterator(_text)), m_scores(_scores) {}

            // get the aligner
            NVBIO_FORCEINLINE NVBIO_HOST_DEVICE const aligner_type &aligner() const { return m_aligner; };

            // return the maximum pattern length
            NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
                uint32
                max_pattern_length() const { return M; }

            // return the maximum text length
            NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
                uint32
                max_text_length() const { return N; }

            // return the stream size
            NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
                uint32
                size() const { return m_count; }

            // return the i-th pattern's length
            NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
                uint32
                pattern_length(const uint32 i, context_type *context) const { return M; }

            // return the i-th text's length
            NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
                uint32
                text_length(const uint32 i, context_type *context) const { return N; }

            // initialize the i-th context
            NVBIO_FORCEINLINE NVBIO_HOST_DEVICE bool init_context(
                const uint32 i,
                context_type *context) const
            {
                context->min_score = Field_traits<int32>::min();
                return true;
            }

            // initialize the i-th context
            NVBIO_FORCEINLINE NVBIO_HOST_DEVICE void load_strings(
                const uint32 i,
                const uint32 window_begin,
                const uint32 window_end,
                const context_type *context,
                strings_type *strings) const
            {
                strings->pattern = pattern_string(M,
                                                  strings->pattern_loader.load(
                                                      m_patterns + i * M,
                                                      M,
                                                      make_uint2(window_begin, window_end),
                                                      false));

                strings->text = text_string(N, strings->text_loader.load(m_text + i * N, N));
            }

            // handle the output
            NVBIO_FORCEINLINE NVBIO_HOST_DEVICE void output(
                const uint32 i,
                const context_type *context) const
            {
                // copy the output score
                m_scores[i] = context->sink.score;
            }

            NVBIO_FORCEINLINE NVBIO_HOST_DEVICE bool test_read_id(
                const uint32 id,
                const context_type *context) const
            {
                return false;
            }

            aligner_type m_aligner;
            uint32 m_count;
            uncached_pattern_iterator m_patterns;
            uncached_text_iterator m_text;
            int16 *m_scores;

            int16 *wfa_H_buffer = nullptr;
            int16 *wfa_H_hi_buffer = nullptr;
            int16 *wfa_H_lo_buffer = nullptr;
            bool  *wfa_H_null_buffer = nullptr;
            int16 *wfa_E_buffer = nullptr;
            int16 *wfa_E_hi_buffer = nullptr;
            int16 *wfa_E_lo_buffer = nullptr;
            bool  *wfa_E_null_buffer = nullptr;
            int16 *wfa_F_buffer = nullptr;
            int16 *wfa_F_hi_buffer = nullptr;
            int16 *wfa_F_lo_buffer = nullptr;
            bool  *wfa_F_null_buffer = nullptr;
            int16 *wfa_PointeurH_buffer = nullptr;
        };

        // A simple kernel to test the speed of alignment without the possible overheads of the BatchAlignmentScore interface
        //
        template <uint32 BLOCKDIM, uint32 MAX_REF_LEN, typename aligner_type, typename score_type>
        __global__ void alignment_test_kernel(const aligner_type aligner, const uint32 N_probs, const uint32 M, const uint32 N, const uint32 *strptr, const uint32 *refptr, score_type *score)
        {
            const uint32 tid = blockIdx.x * BLOCKDIM + threadIdx.x;

            typedef lmem_cache_tag_type lmem_cache_type;
            typedef nvbio::nvbio_cuda::ldg_pointer<uint32> storage_iterator;

            typedef nvbio::PackedStringLoader<storage_iterator, 4, false, lmem_cache_type> pattern_loader_type;
            typedef typename pattern_loader_type::input_iterator uncached_pattern_iterator;
            typedef typename pattern_loader_type::iterator pattern_iterator;
            typedef nvbio::vector_view<pattern_iterator> pattern_string;

            typedef nvbio::PackedStringLoader<storage_iterator, 2, false, lmem_cache_type> text_loader_type;
            typedef typename text_loader_type::input_iterator uncached_text_iterator;
            typedef typename text_loader_type::iterator text_iterator;
            typedef nvbio::vector_view<text_iterator> text_string;

            pattern_loader_type pattern_loader;
            pattern_string pattern = pattern_string(M, pattern_loader.load(uncached_pattern_iterator(strptr) + tid * M, tid < N_probs ? M : 0u));

            text_loader_type text_loader;
            text_string text = text_string(N, text_loader.load(uncached_text_iterator(refptr) + tid * N, tid < N_probs ? N : 0u));

            aln::BestSink<int32> sink;

            aln::alignment_score<MAX_REF_LEN>(
                aligner,
                pattern,
                aln::trivial_quality_string(),
                text,
                Field_traits<int32>::min(),
                sink);

            score[tid] = sink.score;
        }

        //
        // A class for making a single alignment test, testing both scoring and traceback
        //
        struct SingleTest
        {
            thrust::host_vector<uint8> str_hvec;
            thrust::host_vector<uint8> ref_hvec;
            thrust::device_vector<uint8> str_dvec;
            thrust::device_vector<uint8> ref_dvec;
            thrust::device_vector<float> temp_dvec;
            thrust::device_vector<float> score_dvec;
            thrust::device_vector<uint2> sink_dvec;

            // test full DP alignment
            //
            // \param test              test name
            // \param aligner           alignment algorithm
            // \param ref_alignment     reference alignment string
            //
            template <uint32 BLOCKDIM, uint32 N, uint32 M, typename aligner_type>
            void full(const char *test, const aligner_type aligner, const char *ref_alignment)
            {
                NVBIO_VAR_UNUSED const uint32 CHECKPOINTS = 32u;

                typedef ScoreMatrices<N, M, typename aligner_type::aligner_tag> SWMatrices;

                SharedPointer<SWMatrices> mat = SharedPointer<SWMatrices>(new SWMatrices());

                const uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                const uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);

                typename column_storage_type<aligner_type>::type column[N];

                const int32 ref_score = ref_sw<M, N>(str_hptr, ref_hptr, M, N, aligner, mat.get());

                fprintf(stderr, "result=%d\n\n\n", ref_score);
                //mat->print();

                aln::BestSink<int32> sink;
                aln::wfa_type<int32> wfa;
                aln::alignment_score(
                    aligner,
                    vector_view<const uint8 *>(M, str_hptr),
                    trivial_quality_string(),
                    vector_view<const uint8 *>(N, ref_hptr),
                    -1000,
                    sink,
                    column,
                    wfa);

                const int32 cpu_score = sink.score;

                if (cpu_score != ref_score)
                {
                    log_error(stderr, "    expected %s score %d, got: %d\n", test, ref_score, cpu_score);
                    // mat->print();
                    // exit(1);
                }
                else
                {
                    fprintf(stderr, "alignment ok !  score=%d\n\n\n", cpu_score);
                }

                TestBacktracker backtracker;
                backtracker.clear();

                const Alignment<int32> aln = aln::alignment_traceback<1024u, 1024u, CHECKPOINTS>(
                    aligner,
                    vector_view<const uint8 *>(M, str_hptr),
                    trivial_quality_string(),
                    vector_view<const uint8 *>(N, ref_hptr),
                    -1000,
                    backtracker,
                    wfa);

                const int32 aln_score = backtracker.score(aligner, aln.source.x, str_hptr, ref_hptr);
                const std::string aln_string = rle(backtracker.aln).c_str();
                if (aln_score != ref_score)
                {
                    log_error(stderr, "    expected %s backtracking score %d, got %d\n", test, ref_score, aln_score);
                    log_error(stderr, "    %s - %d - [%u, %u] x [%u, %u]\n", aln_string.c_str(), aln.score, aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
                    // mat->print();
                    // exit(1);
                }
                fprintf(stderr, "    %15s : ", test);
                fprintf(stderr, "%d - %s - [%u:%u] x [%u:%u]\n", aln.score, aln_string.c_str(), aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
                if (strcmp(ref_alignment, aln_string.c_str()) != 0)
                {
                    log_error(stderr, "    expected %s, got %s\n", ref_alignment, aln_string.c_str());
                    // mat->print();
                    // exit(1);
                }
            }

            /*template <uint32 BLOCKDIM, uint32 N, uint32 M, typename aligner_type>
            void test_full_wfa(const char *test, const aligner_type aligner, const char *ref_alignment)
            {            
                NVBIO_VAR_UNUSED const uint32 CHECKPOINTS = 248u;
                uint32 NB_TESTS = 500;

                aln::wfa_type<int32> wfa; 

                int16 *wfa_H_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_H_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_H_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_H_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_E_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_E_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_E_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_E_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_F_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_F_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_F_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_F_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_PointeurH_buffer = new int16[WFA_BAND_LEN2_Y];

                wfa.H_Band.set_scores_data(wfa_H_buffer);
                wfa.H_Band.set_lo_data(wfa_H_lo_buffer);
                wfa.H_Band.set_hi_data(wfa_H_hi_buffer);
                wfa.H_Band.set_null_data(wfa_H_null_buffer);
                wfa.E_Band.set_scores_data(wfa_E_buffer);
                wfa.E_Band.set_lo_data(wfa_E_lo_buffer);
                wfa.E_Band.set_hi_data(wfa_E_hi_buffer);
                wfa.E_Band.set_null_data(wfa_E_null_buffer);
                wfa.F_Band.set_scores_data(wfa_F_buffer);
                wfa.F_Band.set_lo_data(wfa_F_lo_buffer);
                wfa.F_Band.set_hi_data(wfa_F_hi_buffer);
                wfa.F_Band.set_null_data(wfa_F_null_buffer);
                wfa.set_pointH_data(wfa_PointeurH_buffer);

                typedef ScoreMatrices<2 * N, 4 * N, typename aligner_type::aligner_tag> SWMatrices;

                SharedPointer<SWMatrices> mat = SharedPointer<SWMatrices>(new SWMatrices());

                const uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                const uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);

                typename column_storage_type<aligner_type>::type column[N];

                int32 ref_score;

                ref_score = -ref_sw<2 * N, 4 * N>(str_hptr, ref_hptr, M, N, aligner, mat.get());
   

                aln::SimpleGotohScheme scoring;
                scoring.m_match = 0;
                scoring.m_mismatch = -2;
                scoring.m_gap_open = -4;
                scoring.m_gap_ext = -1;

                const nvbio::aln::GotohAligner aligner2 = make_gotoh_aligner<aln::SEMI_GLOBAL>(scoring);

                aln::BestSink<int32> sink, sink2;

                Timer timer, timer2;
                timer.start();
   
                for (uint32 i = 0; i < NB_TESTS; i++)
                     aln::banded_alignment_score<31>(
                        aligner2,
                        vector_view<const uint8 *>(M, str_hptr),
                        trivial_quality_string(),
                        vector_view<const uint8 *>(N, ref_hptr),
                        -1000,
                        sink2,
                        wfa);

                timer.stop();

                log_verbose(stderr, "result ref_gotoh:%d\ntime=%f sec\n\n", ref_score, timer.seconds());                

                timer2.start();

                for (uint32 i = 0; i < NB_TESTS; i++)
                {
                    aln::banded_alignment_score<2>(
                        aligner,
                        vector_view<const uint8 *>(M, str_hptr),
                        trivial_quality_string(),
                        vector_view<const uint8 *>(N, ref_hptr),
                        -1000,
                        sink,
                        wfa);

                    const int32 cpu_score = sink.score;

                    if (cpu_score != ref_score)
                    {
                        log_verbose(stderr, "result error !\n\n");
                    }
                }

                timer2.stop();

                log_verbose(stderr, "result ref_wfa=%d\ntime=%f sec\n", sink.score, timer2.seconds());
                log_verbose(stderr, "wfah/gotoh %7.1f x\n\n", sink.score, timer.seconds()/timer2.seconds());
            }*/


            // test full DP alignment
            //
            // \param test              test name
            // \param aligner           alignment algorithm
            // \param ref_alignment     reference alignment string
            //
            template <uint32 BLOCKDIM, uint32 N, uint32 M, typename aligner_type>
            void full_wfa(const char *test, const aligner_type aligner, const char *ref_alignment)
            {
                NVBIO_VAR_UNUSED const uint32 CHECKPOINTS = 10000u;

                typedef ScoreMatrices<2 * N, 4 * N, typename aligner_type::aligner_tag> SWMatrices;

                SharedPointer<SWMatrices> mat = SharedPointer<SWMatrices>(new SWMatrices());

                aln::wfa_type<int32> wfa; 

                int16 *wfa_H_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_H_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_H_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_H_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_E_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_E_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_E_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_E_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_F_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_F_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_F_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_F_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_PointeurH_buffer = new int16[WFA_BAND_LEN2_Y];

                wfa.H_Band.set_scores_data(wfa_H_buffer);
                wfa.H_Band.set_lo_data(wfa_H_lo_buffer);
                wfa.H_Band.set_hi_data(wfa_H_hi_buffer);
                wfa.H_Band.set_null_data(wfa_H_null_buffer);
                wfa.E_Band.set_scores_data(wfa_E_buffer);
                wfa.E_Band.set_lo_data(wfa_E_lo_buffer);
                wfa.E_Band.set_hi_data(wfa_E_hi_buffer);
                wfa.E_Band.set_null_data(wfa_E_null_buffer);
                wfa.F_Band.set_scores_data(wfa_F_buffer);
                wfa.F_Band.set_lo_data(wfa_F_lo_buffer);
                wfa.F_Band.set_hi_data(wfa_F_hi_buffer);
                wfa.F_Band.set_null_data(wfa_F_null_buffer);
                wfa.set_pointH_data(wfa_PointeurH_buffer);

                const uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                const uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);

                typename column_storage_type<aligner_type>::type column[N];

                const int32 ref_score = -ref_sw<2 * N, 4 * N>(str_hptr, ref_hptr, M, N, aligner, mat.get());

                fprintf(stderr, "result=%d\n\n\n", ref_score);
                // mat->print();

                aln::BestSink<int32> sink;
            
                aln::alignment_score(
                    aligner,
                    vector_view<const uint8 *>(M, str_hptr),
                    trivial_quality_string(),
                    vector_view<const uint8 *>(N, ref_hptr),
                    -1000,
                    sink,
                    column,
                    wfa);

                const int32 cpu_score = sink.score;

                if (cpu_score != ref_score)
                {
                    log_error(stderr, "    expected %s score %d, got: %d\n", test, ref_score, cpu_score);
                    // mat->print();
                    // exit(1);
                }
                else
                {
                    log_verbose(stderr, "alignment ok !  score=%d\n\n\n", cpu_score);
                }

                TestBacktracker backtracker;
                backtracker.clear();

                const Alignment<int32> aln = aln::alignment_traceback<1024u, 1024u, CHECKPOINTS>(
                    aligner,
                    vector_view<const uint8 *>(M, str_hptr),
                    trivial_quality_string(),
                    vector_view<const uint8 *>(N, ref_hptr),
                    -1000,
                    backtracker,
                    wfa);

                const int32 aln_score = -backtracker.score(aligner, aln.source.x, str_hptr, ref_hptr);
                const std::string aln_string = rle(backtracker.aln).c_str();
                if (aln_score != ref_score)
                {
                    //log_error(stderr, "    expected %s backtracking score %d, got %d\n", test, ref_score, aln_score);
                    //log_error(stderr, "    %s - %d - [%u, %u] x [%u, %u]\n", aln_string.c_str(), aln.score, aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
                    // mat->print();
                    // exit(1);
                }
                fprintf(stderr, "    %15s : ", test);
                fprintf(stderr, "%d - %s - [%u:%u] x [%u:%u]\n", aln.score, aln_string.c_str(), aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
                if (strcmp(ref_alignment, aln_string.c_str()) != 0)
                {
                    log_error(stderr, "    expected %s, got %s\n", ref_alignment, aln_string.c_str());
                    // mat->print();
                    // exit(1);
                }
                else
                {
                    log_verbose(stderr, "cigar ok !  score=%d  %s\n\n\n", cpu_score, aln_string.c_str());
                }

                delete[] wfa_H_buffer;
                delete[] wfa_H_lo_buffer;
                delete[] wfa_H_hi_buffer;
                delete[] wfa_H_null_buffer;
                delete[] wfa_E_buffer;
                delete[] wfa_E_lo_buffer;
                delete[] wfa_E_hi_buffer;
                delete[] wfa_E_null_buffer;
                delete[] wfa_F_buffer;
                delete[] wfa_F_lo_buffer;
                delete[] wfa_F_hi_buffer;
                delete[] wfa_F_null_buffer;
                delete[] wfa_PointeurH_buffer;
            }

            // test banded alignment
            //
            // \param test              test name
            // \param aligner           alignment algorithm
            // \param ref_alignment     reference alignment string
            //
            template <uint32 BLOCKDIM, uint32 BAND_LEN, const uint32 N, const uint32 M, typename aligner_type>
            void banded(const char *test, const aligner_type aligner, const char *ref_alignment)
            {
                NVBIO_VAR_UNUSED const uint32 CHECKPOINTS = 128u;

                const uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                const uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);

                const int32 ref_score = ref_banded_sw<M, N, BAND_LEN>(str_hptr, ref_hptr, 0u, aligner);

                aln::BestSink<int32> sink;
                aln::wfa_type<int32> wfa;
                aln::banded_alignment_score<BAND_LEN>(
                    aligner,
                    vector_view<const uint8 *>(M, str_hptr),
                    trivial_quality_string(),
                    vector_view<const uint8 *>(N, ref_hptr),
                    -1000,
                    sink,
                    wfa);

                const int32 cpu_score = sink.score;
                if (cpu_score != ref_score)
                {
                    log_error(stderr, "    expected %s score %d, got: %d\n", test, ref_score, cpu_score);
                    //exit(1);
                }

                TestBacktracker backtracker;
                backtracker.clear();

                const Alignment<int32> aln = aln::banded_alignment_traceback<BAND_LEN, 1024u, CHECKPOINTS>(
                    aligner,
                    vector_view<const uint8 *>(M, str_hptr),
                    trivial_quality_string(),
                    vector_view<const uint8 *>(N, ref_hptr),
                    -1000,
                    backtracker,
                    wfa);

                const int32 aln_score = backtracker.score(aligner, aln.source.x, str_hptr, ref_hptr);
                const std::string aln_string = rle(backtracker.aln).c_str();
                if (aln_score != ref_score)
                {
                    log_error(stderr, "    expected %d, got %d\n", ref_score, aln_score);
                    log_error(stderr, "    %s - %d - [%u, %u] x [%u, %u]\n", aln_string.c_str(), aln.score, aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
                    //exit(1);
                }
                fprintf(stderr, "    %15s : ", test);
                fprintf(stderr, "%d - %s - [%u:%u] x [%u:%u]\n", aln.score, aln_string.c_str(), aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
                if (strcmp(ref_alignment, aln_string.c_str()) != 0)
                {
                    log_error(stderr, "    expected %s, got %s\n", ref_alignment, aln_string.c_str());
                    //exit(1);
                }
            }

            template <uint32 BLOCKDIM, uint32 BAND_LEN, const uint32 N, const uint32 M, typename aligner_type>
            void banded_wfa(const char *test, const aligner_type aligner, const char *ref_alignment, int32 score = 0)
            {
                NVBIO_VAR_UNUSED const uint32 CHECKPOINTS = 1000u;

                aln::wfa_type<int32> wfa; 

                int16 *wfa_H_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_H_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_H_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_H_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_E_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_E_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_E_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_E_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_F_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_F_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_F_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_F_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_PointeurH_buffer = new int16[WFA_BAND_LEN2_Y];


                wfa.H_Band.set_scores_data(wfa_H_buffer);
                wfa.H_Band.set_lo_data(wfa_H_lo_buffer);
                wfa.H_Band.set_hi_data(wfa_H_hi_buffer);
                wfa.H_Band.set_null_data(wfa_H_null_buffer);
                wfa.E_Band.set_scores_data(wfa_E_buffer);
                wfa.E_Band.set_lo_data(wfa_E_lo_buffer);
                wfa.E_Band.set_hi_data(wfa_E_hi_buffer);
                wfa.E_Band.set_null_data(wfa_E_null_buffer);
                wfa.F_Band.set_scores_data(wfa_F_buffer);
                wfa.F_Band.set_lo_data(wfa_F_lo_buffer);
                wfa.F_Band.set_hi_data(wfa_F_hi_buffer);
                wfa.F_Band.set_null_data(wfa_F_null_buffer);
                wfa.set_pointH_data(wfa_PointeurH_buffer);

                /*const uint8* str_hptr = nvbio::raw_pointer( str_hvec );
                const uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );

                const int32 ref_score = ref_banded_sw<M,N,BAND_LEN>( str_hptr, ref_hptr, 0u, aligner );*/

                typedef ScoreMatrices<2 * N, 4 * N, typename aligner_type::aligner_tag> SWMatrices;

                SharedPointer<SWMatrices> mat = SharedPointer<SWMatrices>(new SWMatrices());

                const uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                const uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);

                typename column_storage_type<aligner_type>::type column[N];

                int32 ref_score = score;

                if (score == 0)
                    ref_score = -ref_sw<2 * N, 4 * N>(str_hptr, ref_hptr, M, N, aligner, mat.get());

                fprintf(stderr, "result=%d\n\n\n", ref_score);

                aln::BestSink<int32> sink;
                aln::banded_alignment_score<BAND_LEN>(
                    aligner,
                    vector_view<const uint8 *>(M, str_hptr),
                    trivial_quality_string(),
                    vector_view<const uint8 *>(N, ref_hptr),
                    -1000,
                    sink,
                    wfa);

                const int32 cpu_score = sink.score;
                if (cpu_score != ref_score)
                {
                    log_error(stderr, "    expected %s score %d, got: %d\n", test, ref_score, cpu_score);
                    // exit(1);
                }
                else
                {
                    log_verbose(stderr, "alignment ok !  score=%d\n\n\n", cpu_score);
                }

                TestBacktracker backtracker;
                backtracker.clear();

                const Alignment<int32> aln = aln::banded_alignment_traceback<BAND_LEN, 1024u, CHECKPOINTS>(
                    aligner,
                    vector_view<const uint8 *>(M, str_hptr),
                    trivial_quality_string(),
                    vector_view<const uint8 *>(N, ref_hptr),
                    -1000,
                    backtracker,
                    wfa);

                const int32 aln_score = -backtracker.score(aligner, aln.source.x, str_hptr, ref_hptr);
                const std::string aln_string = rle(backtracker.aln).c_str();
                if (aln_score != ref_score)
                {
                    //log_error(stderr, "    expected backtracking score %d, got %d\n", ref_score, aln_score);
                    //log_error(stderr, "    %s - %d - [%u, %u] x [%u, %u]\n", aln_string.c_str(), aln.score, aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
                    // exit(1);
                }
                fprintf(stderr, "    %15s : ", test);
                fprintf(stderr, "%d - %s - [%u:%u] x [%u:%u]\n", aln.score, aln_string.c_str(), aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
                if (strcmp(ref_alignment, aln_string.c_str()) != 0)
                {
                    log_error(stderr, "    expected %s, got %s\n", ref_alignment, aln_string.c_str());
                    // exit(1);
                }
                else
                {
                    log_verbose(stderr, "cigar ok !  score=%d  %s\n\n\n", cpu_score, aln_string.c_str());
                }

                delete[] wfa_H_buffer;
                delete[] wfa_H_lo_buffer;
                delete[] wfa_H_hi_buffer;
                delete[] wfa_H_null_buffer;
                delete[] wfa_E_buffer;
                delete[] wfa_E_lo_buffer;
                delete[] wfa_E_hi_buffer;
                delete[] wfa_E_null_buffer;
                delete[] wfa_F_buffer;
                delete[] wfa_F_lo_buffer;
                delete[] wfa_F_hi_buffer;
                delete[] wfa_F_null_buffer;
                delete[] wfa_PointeurH_buffer;
            }
        };

        // execute a given batch alignment type on a given stream
        //
        // \tparam batch_type               a \ref BatchAlignment "Batch Alignment"
        // \tparam stream_type              a stream compatible to the given batch_type
        //
        // \return                          average time
        //
        template <typename batch_type, typename stream_type>
        float enact_batch(
            batch_type &batch,
            const stream_type &stream,
            const uint32 n_tests,
            const uint32 n_tasks)
        {
            // alloc all the needed temporary storage
            const uint64 temp_size = batch_type::max_temp_storage(
                stream.max_pattern_length(),
                stream.max_text_length(),
                stream.size());

            thrust::device_vector<uint8> temp_dvec(temp_size);

            Timer timer;
            timer.start();

            for (uint32 i = 0; i < n_tests; ++i)
            {
                // enact the batch
                batch.enact(stream, temp_size, nvbio::raw_pointer(temp_dvec));

                hipDeviceSynchronize();
            }

            timer.stop();

            return timer.seconds() / float(n_tests);
        }

        // execute and time a batch of full DP alignments using BatchAlignmentScore
        //
        template <bool supported, typename scheduler_type, uint32 N, uint32 M, typename stream_type>
        struct batch_score_profile_dispatch
        {
            static void run(
                const stream_type stream,
                const uint32 n_tests,
                const uint32 n_tasks)
            {
            }
        };

        // execute and time a batch of full DP alignments using BatchAlignmentScore
        //
        template <typename scheduler_type, uint32 N, uint32 M, typename stream_type>
        struct batch_score_profile_dispatch<true, scheduler_type, N, M, stream_type>
        {
            static void run(
                const stream_type stream,
                const uint32 n_tests,
                const uint32 n_tasks)
            {
                typedef aln::BatchedAlignmentScore<stream_type, scheduler_type> batch_type; // our batch type

                // setup a batch
                batch_type batch;

                const float time = enact_batch(
                    batch,
                    stream,
                    n_tests,
                    n_tasks);

                fprintf(stderr, "  %5.1f", 1.0e-9f * float(n_tasks * uint64(N * M)) / time);
            }
        };

        // execute and time a batch of full DP alignments using BatchAlignmentScore
        //
        template <typename scheduler_type, uint32 N, uint32 M, typename stream_type>
        void batch_score_profile(
            const stream_type stream,
            const uint32 n_tests,
            const uint32 n_tasks)
        {
            NVBIO_VAR_UNUSED const bool is_supported = aln::supports_scheduler<typename stream_type::aligner_type, scheduler_type>::pred;

            batch_score_profile_dispatch<is_supported, scheduler_type, N, M, stream_type>::run(
                stream,
                n_tests,
                n_tasks);
        }

        // execute and time the batch_score<scheduler> algorithm for all possible schedulers
        //
        template <uint32 N, uint32 M, typename aligner_type>
        void batch_score_profile_all(
            const aligner_type aligner,
            const uint32 n_tests,
            const uint32 n_tasks,
            thrust::device_vector<uint32> &pattern_dvec,
            thrust::device_vector<uint32> &text_dvec,
            thrust::device_vector<int16> &score_dvec)
        {
            {
                typedef AlignmentStream<aligner_type, M, N> stream_type;

                 // create a stream
                stream_type stream(
                    aligner,
                    n_tasks,
                    nvbio::raw_pointer(pattern_dvec),
                    nvbio::raw_pointer(text_dvec),
                    nvbio::raw_pointer(score_dvec));

                int16 *wfa_H_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_H_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_H_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_H_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_E_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_E_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_E_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_E_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_F_buffer = new int16[DIM_Y_SHARED];
                int16 *wfa_F_lo_buffer = new int16[WFA_BAND_LEN2_Y];
                int16 *wfa_F_hi_buffer = new int16[WFA_BAND_LEN2_Y];
                bool  *wfa_F_null_buffer = new bool[WFA_BAND_LEN2_Y];
                int16 *wfa_PointeurH_buffer = new int16[WFA_BAND_LEN2_Y];
        

                // Wfa
                stream.wfa_H_buffer = wfa_H_buffer;
                stream.wfa_H_lo_buffer = wfa_H_lo_buffer;
                stream.wfa_H_hi_buffer = wfa_H_hi_buffer;
                stream.wfa_H_null_buffer = wfa_H_null_buffer;
                stream.wfa_E_buffer = wfa_E_buffer;
                stream.wfa_E_lo_buffer = wfa_E_lo_buffer;
                stream.wfa_E_hi_buffer = wfa_E_hi_buffer;
                stream.wfa_E_null_buffer = wfa_E_null_buffer;
                stream.wfa_F_buffer = wfa_F_buffer;
                stream.wfa_F_lo_buffer = wfa_F_lo_buffer;
                stream.wfa_F_hi_buffer = wfa_F_hi_buffer;
                stream.wfa_F_null_buffer = wfa_F_null_buffer;
                stream.wfa_PointeurH_buffer = wfa_PointeurH_buffer;

                // test the DeviceThreadScheduler
                batch_score_profile<DeviceThreadScheduler, N, M>(
                    stream,
                    n_tests,
                    n_tasks);

                // test the DeviceStagedThreadScheduler
                batch_score_profile<DeviceStagedThreadScheduler,N,M>(
                    stream,
                    n_tests,
                    n_tasks );

                delete[] wfa_H_buffer;
                delete[] wfa_H_lo_buffer;
                delete[] wfa_H_hi_buffer;
                delete[] wfa_H_null_buffer;
                delete[] wfa_E_buffer;
                delete[] wfa_E_lo_buffer;
                delete[] wfa_E_hi_buffer;
                delete[] wfa_E_null_buffer;
                delete[] wfa_F_buffer;
                delete[] wfa_F_lo_buffer;                
                delete[] wfa_F_hi_buffer;
                delete[] wfa_F_null_buffer;
                delete[] wfa_PointeurH_buffer;
            }
            /*{
                typedef AlignmentStream<aligner_type,M,N,uncached_tag_type> stream_type;

                // create a stream
                stream_type stream(
                    aligner,
                    n_tasks,
                    nvbio::raw_pointer( pattern_dvec ),
                    nvbio::raw_pointer( text_dvec ),
                    nvbio::raw_pointer( score_dvec ) );

                // test the DeviceWarpScheduler
                batch_score_profile<DeviceWarpScheduler,N,M>(
                    stream,
                    n_tests,
                    n_tasks );
            }
            {
                const uint32 BLOCKDIM = 128;
                const uint32 N_BLOCKS = (n_tasks + BLOCKDIM-1) / BLOCKDIM;

                Timer timer;
                timer.start();

                for (uint32 i = 0; i < n_tests; ++i)
                {
                    // enact the batch
                    alignment_test_kernel<BLOCKDIM,N> <<<N_BLOCKS,BLOCKDIM>>>(
                        aligner,
                        n_tasks,
                        M,
                        N,
                        nvbio::raw_pointer( pattern_dvec ),
                        nvbio::raw_pointer( text_dvec ),
                        nvbio::raw_pointer( score_dvec ) );

                    hipDeviceSynchronize();
                }

                timer.stop();

                const float time = timer.seconds();

                fprintf(stderr,"  %5.1f", 1.0e-9f * float(n_tasks*uint64(N*M))*(float(n_tests)/time) );
            }
            fprintf(stderr, " GCUPS\n");*/
            fprintf(stderr, "\n");
        }

        // execute and time a batch of banded alignments using BatchBandedAlignmentScore
        //
        template <uint32 BAND_LEN, typename scheduler_type, uint32 N, uint32 M, typename stream_type>
        void batch_banded_score_profile(
            const stream_type stream,
            const uint32 n_tests,
            const uint32 n_tasks)
        {
            typedef aln::BatchedBandedAlignmentScore<BAND_LEN, stream_type, scheduler_type> batch_type; // our batch type

            // setup a batch
            batch_type batch;

            const float time = enact_batch(
                batch,
                stream,
                n_tests,
                n_tasks);

            fprintf(stderr, "  %5.2f", 1.0e-9f * float(n_tasks * uint64(BAND_LEN * M)) * (float(n_tests) / time));
        }
        // execute and time the batch_banded_score<scheduler> algorithm for all possible schedulers
        //
        template <uint32 BAND_LEN, uint32 N, uint32 M, typename aligner_type>
        void batch_banded_score_profile_all(
            const aligner_type aligner,
            const uint32 n_tests,
            const uint32 n_tasks,
            thrust::device_vector<uint32> &pattern_dvec,
            thrust::device_vector<uint32> &text_dvec,
            thrust::device_vector<int16> &score_dvec)
        {
            typedef AlignmentStream<aligner_type, M, N> stream_type;

            // create a stream
            stream_type stream(
                aligner,
                n_tasks,
                nvbio::raw_pointer(pattern_dvec),
                nvbio::raw_pointer(text_dvec),
                nvbio::raw_pointer(score_dvec));

            // test the DeviceThreadScheduler
            batch_banded_score_profile<BAND_LEN, DeviceThreadScheduler, N, M>(
                stream,
                n_tests,
                n_tasks);

            // test the DeviceStagedThreadScheduler
            /*batch_banded_score_profile<BAND_LEN, DeviceStagedThreadScheduler, N, M>(
                stream,
                n_tests,
                n_tasks);*/

            // TODO: test DeviceWarpScheduler
            fprintf(stderr, " GCUPS\n");
        }

        // a simple banded edit distance test
        //
        template <typename string_type>
        void banded_edit_distance_test(
            const uint32 test_id,
            const string_type pattern,
            const string_type text,
            const int32 ref_score)
        {
            aln::wfa_type<int32> wfa;

            int16 *wfa_H_buffer = new int16[DIM_Y_SHARED];
            int16 *wfa_H_lo_buffer = new int16[WFA_BAND_LEN2_Y];
            int16 *wfa_H_hi_buffer = new int16[WFA_BAND_LEN2_Y];
            bool *wfa_H_null_buffer = new bool[WFA_BAND_LEN2_Y];
            int16 *wfa_E_buffer = new int16[DIM_Y_SHARED];
            int16 *wfa_E_lo_buffer = new int16[WFA_BAND_LEN2_Y];
            int16 *wfa_E_hi_buffer = new int16[WFA_BAND_LEN2_Y];
            bool *wfa_E_null_buffer = new bool[WFA_BAND_LEN2_Y];
            int16 *wfa_F_buffer = new int16[DIM_Y_SHARED];
            int16 *wfa_F_lo_buffer = new int16[WFA_BAND_LEN2_Y];
            int16 *wfa_F_hi_buffer = new int16[WFA_BAND_LEN2_Y];
            bool *wfa_F_null_buffer = new bool[WFA_BAND_LEN2_Y];
            int16 *wfa_PointeurH_buffer = new int16[WFA_BAND_LEN2_Y];

            wfa.H_Band.set_scores_data(wfa_H_buffer);
            wfa.H_Band.set_lo_data(wfa_H_lo_buffer);
            wfa.H_Band.set_hi_data(wfa_H_hi_buffer);
            wfa.H_Band.set_null_data(wfa_H_null_buffer);
            wfa.E_Band.set_scores_data(wfa_E_buffer);
            wfa.E_Band.set_lo_data(wfa_E_lo_buffer);
            wfa.E_Band.set_hi_data(wfa_E_hi_buffer);
            wfa.E_Band.set_null_data(wfa_E_null_buffer);
            wfa.F_Band.set_scores_data(wfa_F_buffer);
            wfa.F_Band.set_lo_data(wfa_F_lo_buffer);
            wfa.F_Band.set_hi_data(wfa_F_hi_buffer);
            wfa.F_Band.set_null_data(wfa_F_null_buffer);
            wfa.set_pointH_data(wfa_PointeurH_buffer);

            const int32 ed = banded_alignment_score<5>(
                make_edit_distance_aligner<aln::SEMI_GLOBAL>(),
                pattern,
                text,
                -255,
                wfa);

            if (ed != ref_score)
            {
                log_error(stderr, "  synthetic Edit Distance test %u... failed\n", test_id);
                log_error(stderr, "    expected %d, got: %d - pattern: %s text: %s\n", ref_score, ed, pattern.begin(), text.begin());
                exit(1);
            }
            else
                fprintf(stderr, "  synthetic Edit Distance test %u... passed!\n", test_id);

            delete[] wfa_H_buffer;
            delete[] wfa_H_lo_buffer;
            delete[] wfa_H_hi_buffer;
            delete[] wfa_H_null_buffer;
            delete[] wfa_E_buffer;
            delete[] wfa_E_lo_buffer;
            delete[] wfa_E_hi_buffer;
            delete[] wfa_E_null_buffer;
            delete[] wfa_F_buffer;
            delete[] wfa_F_lo_buffer;
            delete[] wfa_F_hi_buffer;
            delete[] wfa_F_null_buffer;
            delete[] wfa_PointeurH_buffer;
        }

        void test(int argc, char *argv[])
        {
            uint32 n_tests = 1;
            NVBIO_VAR_UNUSED uint32 N_WARP_TASKS = 4096;
            uint32 N_THREAD_TASKS = 2 * 1024;
            uint32 TEST_MASK = 0xFFFFFFFFu;

            for (int i = 0; i < argc; ++i)
            {
                if (strcmp(argv[i], "-N-thread-tasks") == 0)
                    N_THREAD_TASKS = atoi(argv[++i]);
                else if (strcmp(argv[i], "-N-warp-tasks") == 0)
                    N_WARP_TASKS = atoi(argv[++i]);
                else if (strcmp(argv[i], "-N-tests") == 0)
                    n_tests = atoi(argv[++i]);
                else if (strcmp(argv[i], "-tests") == 0)
                {
                    const std::string tests_string(argv[++i]);

                    char temp[256];
                    const char *begin = tests_string.c_str();
                    const char *end = begin;

                    TEST_MASK = 0u;

                    while (1)
                    {
                        while (*end != ':' && *end != '\0')
                        {
                            temp[end - begin] = *end;
                            end++;
                        }

                        temp[end - begin] = '\0';

                        if (strcmp(temp, "functional") == 0)
                            TEST_MASK |= FUNCTIONAL;
                        else if (strcmp(temp, "ed") == 0)
                            TEST_MASK |= ED;
                        else if (strcmp(temp, "ed-banded") == 0)
                            TEST_MASK |= ED_BANDED;
                        else if (strcmp(temp, "sw") == 0)
                            TEST_MASK |= SW;
                        else if (strcmp(temp, "sw-banded") == 0)
                            TEST_MASK |= SW_BANDED;
                        else if (strcmp(temp, "sw-warp") == 0)
                            TEST_MASK |= SW_WARP;
                        else if (strcmp(temp, "sw-striped") == 0)
                            TEST_MASK |= SW_STRIPED;
                        else if (strcmp(temp, "gotoh") == 0)
                            TEST_MASK |= GOTOH;
                        else if (strcmp(temp, "gotoh-banded") == 0)
                            TEST_MASK |= GOTOH_BANDED;
                        else if (strcmp(temp, "wfa") == 0)
                            TEST_MASK |= WFA;
                        else if (strcmp(temp, "wfa-banded") == 0)
                            TEST_MASK |= WFA_BANDED;

                        if (*end == '\0')
                            break;

                        ++end;
                        begin = end;
                    }
                }
            }

            fprintf(stderr, "testing alignment... started\n");

            if (TEST_MASK & FUNCTIONAL)
            {
                typedef vector_view<const char *> const_string;

                // right aligned, no gaps
                {
                    const_string text = make_string("AAAAGGGTGCTCAA");
                    const_string pattern = make_string("GGGTGCTCAA");

                    banded_edit_distance_test(
                        1u,      // test id
                        pattern, // pattern
                        text,    // text
                        0);      // expected score
                }
                // right aligned, 2 insertions
                {
                    const_string text = make_string("AAAAGGGTGCTCAA");
                    const_string pattern = make_string("GGGTAAGCTC");

                    banded_edit_distance_test(
                        2u,      // test id
                        pattern, // pattern
                        text,    // text
                        -2);     // expected score
                }
                // right aligned, 2 deletions
                {
                    const_string text = make_string("AAAAGGGTGCAATC");
                    const_string pattern = make_string("AAGGGTGCTC");

                    banded_edit_distance_test(
                        3u,      // test id
                        pattern, // pattern
                        text,    // text
                        -2);     // expected score
                }
                // left aligned, zero gaps
                {
                    const_string text = make_string("AAAAGGGTGCTCAA");
                    const_string pattern = make_string("AAAAGGGTGC");

                    banded_edit_distance_test(
                        4u,      // test id
                        pattern, // pattern
                        text,    // text
                        0);      // expected score
                }
                // left aligned, 2 deletions
                {
                    const_string text = make_string("AAAAGGAAGTGCTC");
                    const_string pattern = make_string("AAAAGGGTG");

                    banded_edit_distance_test(
                        5u,      // test id
                        pattern, // pattern
                        text,    // text
                        -2);     // expected score
                }
                // centrally aligned, 2 insertions
                {
                    const_string text = make_string("AACAGGGTGCTC");
                    const_string pattern = make_string("CACCGGGT");

                    banded_edit_distance_test(
                        6u,      // test id
                        pattern, // pattern
                        text,    // text
                        -2);     // expected score
                }
            }
           
            bool unit_tests = true;
            bool test1 = true;
            bool test2 = true;
            //bool test3 = false;

            if (false && test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;
                const uint32 M = 8;
                const uint32 N = 8+7; 

                thrust::host_vector<uint8> str_hvec(M);
                thrust::host_vector<uint8> ref_hvec(N);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);

                string_to_dna("ACATGACA", str_hptr);
                string_to_dna("AACATTGATGACACA", ref_hptr);

                // In the direction <- 4M 1D 3M
                // AAACACCCTAACACACTAAA
                //           ACA ACTA

                // In the direction <- 1M 2D 3M 1D 3M 10D
                //  AACACCCTAACACACTAAA
                //           ACA ACT  A

                // In the direction <- 2M 1D 3M 11D 3M
                //  AAACACCCTAACACACTAAA
                //  ACA           ACT AA

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);
                
                {
                    fprintf(stderr, "  testing Gotoh scoring...\n");
                    aln::SimpleGotohScheme scoring;
                    scoring.m_match = 0;     // 2;
                    scoring.m_mismatch = -7; //-4;//-1;
                    scoring.m_gap_open = -4; //-6;//-1;
                    scoring.m_gap_ext = -1;  //-1;//-1;

                    //test.full<BLOCKDIM, N, M>("global", make_gotoh_aligner<aln::GLOBAL>(scoring), "5M11D3M");
                    // test.full<BLOCKDIM,N,M>(       "local", make_gotoh_aligner<aln::LOCAL>( scoring ),       "4M1D3M" );
                    // test.full<BLOCKDIM,N,M>( "semi-global", make_gotoh_aligner<aln::SEMI_GLOBAL>( scoring ), "4M1D3M" );
                    test.banded<BLOCKDIM, 7u, N, M>( "global", make_gotoh_aligner<aln::SEMI_GLOBAL>( scoring ), "6M4D2M" );                                    }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char str[] = "ACATGACA";
                char ref[] = "AACATTGATGACACA";
                
                const uint32 M = 8;
                const uint32 N = 8+7; 

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "2D7M5D1M");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "3M1I4M1I" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 15u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "2D7M5D1M");
                }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char str[] = "ACGGGACA";
                char ref[] = "AACATTGATGACACA";
                
                const uint32 M = 8;
                const uint32 N = 8+7; 

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "7M7D1M");
                    //test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "3M1I4M1I" );
                    //test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 15u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "7M7D1M");
                }
            }

            if (false && test1 && TEST_MASK & FUNCTIONAL)
            {
                fprintf(stderr,"  testing real banded Gotoh problem...\n");
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;
                NVBIO_VAR_UNUSED const uint32 BAND_LEN = 31;
                NVBIO_VAR_UNUSED const uint32 M = 150;
                NVBIO_VAR_UNUSED const uint32 N = 150 + 31;

                thrust::host_vector<uint8> str_hvec( M );
                thrust::host_vector<uint8> ref_hvec( N );

                uint8* str_hptr = nvbio::raw_pointer( str_hvec );
                uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );
                string_to_dna("TTATGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTAT", str_hptr);
                string_to_dna("ATCGGATTCTTTCTTACTTGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTATCTCTCTCTCCATCTAT", ref_hptr);

                aln::SimpleGotohScheme scoring;
                scoring.m_match    =  0;
                scoring.m_mismatch = -2;
                scoring.m_gap_open = -4;
                scoring.m_gap_ext  = -1;

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                test.banded<BLOCKDIM, BAND_LEN, N, M>( "global", make_gotoh_aligner<aln::SEMI_GLOBAL>( scoring ), "147M2D3M" );
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char str[] = "TTATGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTAT";
                char ref[] = "ATCGGATTCTTTCTTACTTGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTATCTCTCTCTCCATCTAT";
                
                NVBIO_VAR_UNUSED const uint32 M = 150;
                NVBIO_VAR_UNUSED const uint32 N = 150 + 31;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "16D148M15D2M");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "3M1I4M1I" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 181u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "16D148M15D2M");
                }
            }


            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char str[] = "AGGATCGG";     // 8
                char ref[] = "AACCATACTCGG"; // 12

                const uint32 M = 8;
                const uint32 N = 12;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "7M4D1M");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 151u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "7M4D1M");
                }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char str[] = "ACAACTA";
                char ref[] = "GAACACCCTAACACACTAAG";

                const uint32 M = 7;
                const uint32 N = 20;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "2D4M11D3M");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 20u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "2D4M11D3M");
                }
            }

            if (unit_tests && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char str[] = "ACCCGAA";
                char ref[] = "GAACACCCTAACACACTAAG";

                const uint32 M = 7;
                const uint32 N = 20;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "9D7M4D");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 20u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "9D7M4D");
                }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char ref[] = "AAACACCCTAACACACTAA";
                char str[] = "AAACATAACACACTAA";

                const uint32 M = 16;
                const uint32 N = 19;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "11M3D5M");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 19u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "11M3D5M");
                }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char ref[] = "ATCGGATTCTTTCTTACTTGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTATCTCTCTCTCCATCTAT";
                char str[] = "TATGTAGT";

                const uint32 M = 8;
                const uint32 N = 150 + 31;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "153D8M20D");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 181u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "153D8M20D");
                }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char ref[] = "ATCGGATTCTTTCTTACTTGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTATCTCTCTCTCCATCTAT";
                char str[] = "TTATGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTAT";

                const uint32 M = 150;
                const uint32 N = 150 + 31;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "16D148M15D2M");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 181u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "16D148M15D2M");
                }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char ref[] = "CATCAATAGAAAACCGGATTAAAAACATCGAAAATTATTGAAAAAATATTAAGTGTAGTGTGGAAATGAATGAGTAGAAAAAAAGATAAATTAGAAAACAGAACATCAACTTCGTAAATAGTAAAACGCTAAGCCAGACTAGGTAGAACTAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAA";
                char str[] = "GAGCTAAGAAGGCGCTCATCAATAGAAAACCGGATTAAAAACATCGAAAATTATTGAAAAAATATTAAGTGTAGTGTGGAAATGAATGAGTAGAAAAAAGATAAATTAGAAAACAGAACATCAACTTCGTAAATAGTAAAACGCTAAGCC";

                const uint32 M = 150;
                const uint32 N = 181;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "46D51M1D83M16I");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 181u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "46D51M1D83M16I");
                }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char ref[] = "CTAAATACGAAACCCACACTCGTTTTAATTCAAATCTCATAACCATAAAAAAAAAGCACAATTCAACTTGAGCACGCACACTAAGTAGTAACAACGTTCATTTACAGTAAAGCGAACGGACGAAACAAATAAAAGAAAGGCATAGTGAGNAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAA";
                char str[] = "TTTACGGGATCTGTCTAAATACGAAACACACACTCGTTTTAGTTCAAATATCATAACCATAAAAAAAAAAGCACAATTCAACTTGAGCACGCACACTAAGTAGTAACAACGTTCATTTACAGTAAAGCGAACGGACGAAACAAATAAAAG";

                int a = strlen(ref);
                int b = strlen(str);
                const uint32 M = 150;
                const uint32 N = 181;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "46D80M1I55M14I");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 181u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "46D80M1I55M14I");
                }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char ref[] = "TGCTGTTGTTGCTGTTGCTGTTGTTGCTGTTGCTGTTGTTGCTGTTGTTGGAGATGATTCTGAGTGTAGTGAGCCTGAAATTCCATTTTATCAAAACGCTGAGGAATATTCATCGATCCCTGCATGCTAGTAGGGGTTGTCATAACGCTATTGATCGAGCTCAAAGGCATTTTTTGTTGTT";
                char str[] = "TGCTGTTGTTGCTGTTGCTGTTGTTGCTGTTGCTGTTGTTGCTGTTGTTGGAGATGATTCTGAGTGTAGTGAGCCTGAAATTCCATTTTATCAAAACGCTGAGGAATATTCATCGATCCCTGCATGCTAGTAGGGGTTGTCATAACGCTA";

                int a = strlen(ref);
                int b = strlen(str);
                const uint32 M = 150;
                const uint32 N = 181;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "31D150M");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 181u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "31D150M");
                }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char ref[] = "TAGTGGGTGACCATACGCGAAACTCAGGTGCTGCAATCTTTTTTTTTTTTCCGCGCGCAAGCACGTTACCCGGACCCCGTCTTAGCACACGCACACGCACACGCAGCGCTCACAGACCAGCGAAACAGACCTGAGAGCCACGATGCAGCACACGCTTACCCGGACCGCCTCTCTGCCAGAA";
                char str[] = "NGCGAAACTCAGGTGCTGCAATCTTTATTTCTTTTTTTTTTTTTTTTTGTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTGTGTGTTTGTTTGGGTTGTTTTTTTTTTTAATTTT";

                int a = strlen(ref);
                int b = strlen(str);
                const uint32 M = 150;
                const uint32 N = 181;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "1M3I71M26D42M7I26M15D");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 181u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "1M3I71M26D42M7I26M15D");

                    //return;
                }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 1;

                char ref[] = "ATGATGAGACACCTGTTTTTGGTCAGGATCAAAATACCACGAAGTCCAAGGTTGTTCAATTGATTGGCGCCGTACAGACATTACTGAGGAGTATGTTATGTTGATGGAGAACGGTTAAAGTTACATTTCATCAGTTTTTTCCCGTTCTTTTTCACCTTTTGTGAGAAAATTTTACTAACGT";                         
                char str[] = "TTTTTGGTCAGGATCAAAATACCACGAAGTCCAAGGTTGTTCAATTGATTGGCGCCGTACAGACATTACTGAGGATACCATTAATTGGAATAAATATACTGGTGATTGTTTATGAATTGCTATTGGGATGAACTAAGCGTACAAAGCAAA";

                const uint32 M = 150;
                const uint32 N = 181;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 1;
                    scoring.m_gap_open = 1;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM,N,M>(      "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3D51M3D9M4D3M5D12M1D75M15D" );
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 2, N, M>("semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "3D51M3D9M4D3M5D12M1D75M15D");
                }
            }

            if (test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char ref[] = "TCGTCCTTGTCCAAAAAGTCTTGATAACATTAAAAGTCACTACCGTCAAATCCATCATCAAATCCGCCGTCGAACCCACCAGCATCATCAAATCCGCCGTCGGACCCACCAGCATCATCACCGTAGTAATTGTTCTCGACAACGACAGTGTCTGGTCCGTCATAGTTGTGGTCGTCAAATG";
                char str[] = "GCTTATATCATTTTATCGTCCTTGCCCAAAAAGTCTTGATAACATTAAAAGTCACCACCGTCAAATCCATCATCAAATCCGCCGTCGAACCCACCAGCATCATCACCGTAGTAATTGTTCTCGACAACGACAGTGTCTGGTTCGTCATAG";

                int a = strlen(ref);
                int b = strlen(str);
                const uint32 M = 150;
                const uint32 N = 181;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 1;
                    scoring.m_gap_open = 1;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "16D45M30D90M15I");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 181u, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "16D45M30D90M15I");

                    //return;
                }
            }

            if (test2 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 100;

                char ref[] = "GCCGACCTCTGTTTCGGCATCGGGCAAGAAGATCTTGACACCATTTCTGGCAGCTGGCCCCTTTTTCAAGTATTTGAATCCGACTCGTACCTTGCTATACGTTTTATTGTTTAGCTGGTCCATTATCTTGGCATAGCCATTGAACCAGTACTTTTGATCTACTAGGTCCCTTCTTGACTTTGAAATCACCCAGTTTAACGCAGCTTCTACTGGTGTGA"
                             "TACTTTCGTCCAATTCATGACCATACAAACACATACCAGCTTCCAACCTTAAACTGTCTCTAGCAGCCAGTCCGATAGGCTTCATTACTGGATTGGCCAAGAGTTGCTCCGCAAACTCAACCGCTTTCTCATTTGCAATGCTTATCTCAAATCCATCTTCACCAGTGTACCCGCCTCTAGCAATTTGAACCAAAGAACCGTCCTTTAACGCAAATTCA"
                             "TGTCTTTGTCCAAAAAATAACTCTTTTAGATCCTTTCCAGGAGCTGTTTTTGATAAAAGTGGTT";
                char str[] = "TGTTTAGCTGGTCCATTATCTTGGCATAGCCATTGAACCAGTACTTTTGATCTACTAGGTCCCTTCTTGACTTTGAAATCACCCAGTTTAACGCAGCTTCTACTGGTGTGATACTTTCGTCCAATTCATGACCATACAAACACATACCAG";

                const uint32 M = 150;
                const uint32 N = 500;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 1;
                    scoring.m_gap_open = 1;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM,N,M>(      "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "243D150M107D" );
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 501, N, M>("semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "243D150M107D");
                }
            }

            if (false && test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char ref[] = "CAGCACTGACCGGTGAGCATAAACCCTGGGGATGCCCAGAGCTGGTACAGCCAGGAGCTCCAGAAGCGTGGGATTCTCAGAGGGAAGTGGAGCTCACTGCTCTACAGGTCCTATTCAAGTTAGAAAGTAAGATACAATGCACACAAAGCCAAATTGTC"
                             "ATCATTCAGCTCCTATTACAGGGGAACTAAGAGCTGCATTGAAAATTATTTGCAAAGCTTGTAAGTGGTTCTGCCACTTATTAGCCGTGTGAACCTTAGCAAATTACCTAGCGTCTCTGAGTTTCAACTTCCTCATCTACAAAATAGAAATGATAATAAT"
                             "AACCGCATCGCAAGAGTTGTTGGAAAAATGAAAATGAGGTATCATAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAACAGCACCTGGT"
                             "AAATTAAGTATTGAAAAAATGCCAGCACTGACCGGTGAGCATAAACCCTGGGGATGCCCAGAGCTGGTACAGCCAGGAGCTCCAGAAGCGTGGGATTCTCAGAGGGAAGTGGAGCTCACTGCTCTACAGGTCCTATTCAAGTTAGAAAGTAAGATACAATGCACACAAAGCCAAATTGTC"
                             "ATCATTCAGCTCCTATTACAGGGGAACTAAGAGCTGCATTGAAAATTATTTGCAAAGCTTGTAAGTGGTTCTGCCACTTATTAGCCGTGTGAACCTTAGCAAATTACCTAGCGTCTCTGAGTTTCAACTTCCTCATCTACAAAATAGAAATGATAATAAT"
                             "AACCGCATCGCAAGAGTTGTTGGAAAAATGAAAATGAGGTATCATAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAACAGCACCTGGT"
                             "AAATTAAGTATTGAAAAAATGC";
                char str[] = "TAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAGCAGCACCTGGTAAATTAAGTATTGAAAAAATGCAGATCG";

                const uint32 M = 144;
                const uint32 N = 1000;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM,N,M>(      "global", make_wfah_aligner<aln::GLOBAL>( scoring ), "6I138M362D" );
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "5M12D3M" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 181, N, M>("global", make_wfah_aligner<aln::GLOBAL>(scoring), "6I138M362D", -110);
                }
            }

            if (false && test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char str[] = "TTATGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTAT";
                char ref[] = "CGACTGACGTGGTATCTCTCTCTCCATCTATTTATGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTAT";
                
                NVBIO_VAR_UNUSED const uint32 M = 150;
                NVBIO_VAR_UNUSED const uint32 N = 150 + 31;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "150M31D");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "3M1I4M1I" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 181, N, M>("global", make_wfah_aligner<aln::GLOBAL>(scoring), "150M31D");
                }
            }

            if (false && test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char str[] = "TTATGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTAT";
                char ref[] = "CGACTGACGTGGTATCTCTCTCTCCATCTATTTATGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTATCGACTGACGTGGTATCT";
                
                NVBIO_VAR_UNUSED const uint32 M = 150;
                NVBIO_VAR_UNUSED const uint32 N = 150 + 31;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "150M31D");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "3M1I4M1I" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 181, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "150M31D");
                }
            }

            if (TEST_MASK & FUNCTIONAL)
            {
                fprintf(stderr,"  testing real banded Gotoh problem...\n");
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;
                NVBIO_VAR_UNUSED const uint32 BAND_LEN = 31;
                NVBIO_VAR_UNUSED const uint32 M = 150;
                NVBIO_VAR_UNUSED const uint32 N = 150 + 31;

                thrust::host_vector<uint8> str_hvec( M );
                thrust::host_vector<uint8> ref_hvec( N );

                uint8* str_hptr = nvbio::raw_pointer( str_hvec );
                uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );
                string_to_dna("TTATGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTAT", str_hptr);
                string_to_dna("ATCGGATTCTTTCTTACTTGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTATCTCTCTCTCCATCTAT", ref_hptr);

                aln::SimpleGotohScheme scoring;
                scoring.m_match    =  0;
                scoring.m_mismatch = -5;
                scoring.m_gap_open = -8;
                scoring.m_gap_ext  = -3;

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                test.banded<BLOCKDIM, BAND_LEN, N, M>( "banded-semi-global", make_gotoh_aligner<aln::SEMI_GLOBAL>( scoring ), "147M2D3M" );
            }

            /*if (false && test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char str[] = "TTATGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTAT";
                char ref[] = "ATCGGATTCTTTCTTACTTGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTATCTCTCTCTCCATCTAT";
                
                NVBIO_VAR_UNUSED const uint32 M = 150;
                NVBIO_VAR_UNUSED const uint32 N = 150 + 31;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "\n\ntesting Wfa scoring... (match:%i mismatch:%i gap_open:%i gap_ext:%i)\nref:%s\nstr:%s\n", scoring.m_match, scoring.m_mismatch, scoring.m_gap_open, scoring.m_gap_ext, ref, str);

                    //test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::GLOBAL>(scoring), "5M11D3M");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "3M1I4M1I" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.test_full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::SEMI_GLOBAL>(scoring), "16D148M15D2M");
                }
            }*/

            // Tests notation

            if (false && test1 && TEST_MASK & FUNCTIONAL)
            {
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;

                char str[] = "TGACGTAGGGACTGTCATGTCAATGCTAAGTGGTTCTGGCGGCGGGAGCCAAAGTATGGGTGCTTCCGGCCTGGCTGCCTTGGCTTCTCAATTCTTTAAGTCAGGTAACAATTCCCAAGGTCAGGGACAAGGTCAAGGTCAAGGTCAAGGTCAAGGACAAGGTCAAGGTCAAGGTTCTTTTACTGCTTTGGCGTCTTTGGCTTCATCTTTCATGAATTCCAACAACAATAATCAGCAAGGTCAAAATCAAAGCTCCGGTGGTTCCTCCTTTGGAGCACTGGCTTCTATGGCAAGCTCTTTTATGCATTCCAATAATAATCAGAACTCCAACAATAGTCAACAGGGCTATAACCAATCCTATCAAAACGGTAACCAAAATAGTCAAGGTTACAATAATCAACAGTACCAAGGTGGCAACGGTGGTTACCAACAACAACAGGGACAATCTGGTGGTGCTTTTTCCTCATTGGCCTCCATGGCTCAATCTTACTTAGGTGGTG";
                char ref[] = "CAACAATAGTCAACAGGGCTATAACCAATCCTATCAAAACGGTAACCAAAATAGTCAAGGTTACAATAATCAACAGTACCAAGGTGGCAACGGTGGTTACCAACAACAACAGGGACAATCTGGTGGTGCTTTTTCCCCATTGGCCTCCAT";
                
                NVBIO_VAR_UNUSED const uint32 M = 150;
                NVBIO_VAR_UNUSED const uint32 N = 500;

                thrust::host_vector<uint8> ref_hvec(N);
                thrust::host_vector<uint8> str_hvec(M);

                uint8 *str_hptr = nvbio::raw_pointer(str_hvec);
                uint8 *ref_hptr = nvbio::raw_pointer(ref_hvec);
                string_to_dna(ref, ref_hptr);
                string_to_dna(str, str_hptr);

                SingleTest test;
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::nvbio_cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                {
                    fprintf(stderr, "  testing Wfa scoring...\n");
                    aln::SimpleGotohScheme scoring2;
                    scoring2.m_match = 0;
                    scoring2.m_mismatch = -1;
                    scoring2.m_gap_open = -1;
                    scoring2.m_gap_ext = -1;

                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 1;
                    scoring.m_gap_open = 1;
                    scoring.m_gap_ext = 1;

                    test.full<BLOCKDIM, N, M>("global", make_gotoh_aligner<aln::GLOBAL>(scoring2), "31D150M");
                    test.full_wfa<BLOCKDIM, N, M>("global", make_wfah_aligner<aln::GLOBAL>(scoring), "31D150M");
                    // test.full_wfa<BLOCKDIM,N,M>(       "local", make_wfah_aligner<aln::LOCAL>( scoring ), "3M1I4M1I" );
                    // test.full_wfa<BLOCKDIM,N,M>( "semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "3M1I4M1I" );
                    test.banded_wfa<BLOCKDIM, 2, N, M>("global", make_wfah_aligner<aln::GLOBAL>(scoring), "31D150M");
                }
            }

            // This code is for debugging purposes, useful to plug-in and analyze real problems coming from an app
            /*if (false && TEST_MASK & FUNCTIONAL)
            {
                fprintf(stderr,"  testing real full-matrix Gotoh problem...\n");
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;
                NVBIO_VAR_UNUSED const uint32 M = 144;
                NVBIO_VAR_UNUSED const uint32 N = 500;

                thrust::host_vector<uint8> str_hvec( M );
                thrust::host_vector<uint8> ref_hvec( N );

                uint8* str_hptr = nvbio::raw_pointer( str_hvec );
                uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );

                const char* str_ascii =
                    "TAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAGCAGCACCTGGTAAATTAAGTATTGAAAAAATGCAGATCG";
                const char* ref_ascii =
                    "CAGCACTGACCGGTGAGCATAAACCCTGGGGATGCCCAGAGCTGGTACAGCCAGGAGCTCCAGAAGCGTGGGATTCTCAGAGGGAAGTGGAGCTCACTGCTCTACAGGTCCTATTCAAGTTAGAAAGTAAGATACAATGCACACAAAGCCAAATTGTC"
                    "ATCATTCAGCTCCTATTACAGGGGAACTAAGAGCTGCATTGAAAATTATTTGCAAAGCTTGTAAGTGGTTCTGCCACTTATTAGCCGTGTGAACCTTAGCAAATTACCTAGCGTCTCTGAGTTTCAACTTCCTCATCTACAAAATAGAAATGATAATAAT"
                    "AACCGCATCGCAAGAGTTGTTGGAAAAATGAAAATGAGGTATCATAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAACAGCACCTGGT"
                    "AAATTAAGTATTGAAAAAATGC";

                string_to_dna( str_ascii, str_hptr );
                string_to_dna( ref_ascii, ref_hptr );

                aln::SimpleGotohScheme scoring;
                scoring.m_match    =  0;
                scoring.m_mismatch = -5;
                scoring.m_gap_open = -8;
                scoring.m_gap_ext  = -3;

                aln::GotohAligner<aln::SEMI_GLOBAL, aln::SimpleGotohScheme> aligner( scoring );

                SingleTest test;
                nvbio::cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                test.full<BLOCKDIM,N,M>( "semi-global", aligner, "6I138M" );
            }


            if (false && TEST_MASK & FUNCTIONAL)
            {
                fprintf(stderr,"  testing real banded Wfa problem...\n");
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;
                NVBIO_VAR_UNUSED const uint32 BAND_LEN = 31;
                NVBIO_VAR_UNUSED const uint32 M = 150;
                NVBIO_VAR_UNUSED const uint32 N = 150 + 31;

                thrust::host_vector<uint8> str_hvec( M );
                thrust::host_vector<uint8> ref_hvec( N );

                uint8* str_hptr = nvbio::raw_pointer( str_hvec );
                uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );
                string_to_dna("TTATGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTAT", str_hptr);
                string_to_dna("ATCGGATTCTTTCTTACTTGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTATCTCTCTCTCCATCTAT", ref_hptr);

                aln::SimpleWfahScheme scoring;
                scoring.m_match    =  0;
                scoring.m_mismatch = -5;
                scoring.m_gap_open = -8;
                scoring.m_gap_ext  = -3;

                SingleTest test;
                nvbio::cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                test.banded<BLOCKDIM, BAND_LEN, N, M>( "banded-semi-global", make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ), "147M2D3M" );
            }

            // This code is for debugging purposes, useful to plug-in and analyze real problems coming from an app
            if (false && TEST_MASK & FUNCTIONAL)
            {
                fprintf(stderr,"  testing real full-matrix Wfa problem...\n");
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;
                NVBIO_VAR_UNUSED const uint32 M = 144;
                NVBIO_VAR_UNUSED const uint32 N = 500;

                thrust::host_vector<uint8> str_hvec( M );
                thrust::host_vector<uint8> ref_hvec( N );

                uint8* str_hptr = nvbio::raw_pointer( str_hvec );
                uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );

                const char* str_ascii =
                    "TAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAGCAGCACCTGGTAAATTAAGTATTGAAAAAATGCAGATCG";
                const char* ref_ascii =
                    "CAGCACTGACCGGTGAGCATAAACCCTGGGGATGCCCAGAGCTGGTACAGCCAGGAGCTCCAGAAGCGTGGGATTCTCAGAGGGAAGTGGAGCTCACTGCTCTACAGGTCCTATTCAAGTTAGAAAGTAAGATACAATGCACACAAAGCCAAATTGTC"
                    "ATCATTCAGCTCCTATTACAGGGGAACTAAGAGCTGCATTGAAAATTATTTGCAAAGCTTGTAAGTGGTTCTGCCACTTATTAGCCGTGTGAACCTTAGCAAATTACCTAGCGTCTCTGAGTTTCAACTTCCTCATCTACAAAATAGAAATGATAATAAT"
                    "AACCGCATCGCAAGAGTTGTTGGAAAAATGAAAATGAGGTATCATAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAACAGCACCTGGT"
                    "AAATTAAGTATTGAAAAAATGC";

                string_to_dna( str_ascii, str_hptr );
                string_to_dna( ref_ascii, ref_hptr );

                aln::SimpleWfahScheme scoring;
                scoring.m_match    =  0;
                scoring.m_mismatch = -5;
                scoring.m_gap_open = -8;
                scoring.m_gap_ext  = -3;

                aln::WfahAligner<aln::SEMI_GLOBAL, aln::SimpleWfahScheme> aligner( scoring );

                SingleTest test;
                nvbio::cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                test.full<BLOCKDIM,N,M>( "semi-global", aligner, "6I138M" );
            }*/


            // This code is for debugging purposes, useful to plug-in and analyze real problems coming from an app
            /*if (TEST_MASK & FUNCTIONAL)
            {
                fprintf(stderr,"  testing real full-matrix Edit Distance problem...\n");
                NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;
                NVBIO_VAR_UNUSED const uint32 M = 144;
                NVBIO_VAR_UNUSED const uint32 N = 500;

                thrust::host_vector<uint8> str_hvec( M );
                thrust::host_vector<uint8> ref_hvec( N );

                uint8* str_hptr = nvbio::raw_pointer( str_hvec );
                uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );

                const char* str_ascii =
                    "TAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAGCAGCACCTGGTAAATTAAGTATTGAAAAAATGCAGATCG";
                const char* ref_ascii =
                    "CAGCACTGACCGGTGAGCATAAACCCTGGGGATGCCCAGAGCTGGTACAGCCAGGAGCTCCAGAAGCGTGGGATTCTCAGAGGGAAGTGGAGCTCACTGCTCTACAGGTCCTATTCAAGTTAGAAAGTAAGATACAATGCACACAAAGCCAAATTGTC"
                    "ATCATTCAGCTCCTATTACAGGGGAACTAAGAGCTGCATTGAAAATTATTTGCAAAGCTTGTAAGTGGTTCTGCCACTTATTAGCCGTGTGAACCTTAGCAAATTACCTAGCGTCTCTGAGTTTCAACTTCCTCATCTACAAAATAGAAATGATAATAAT"
                    "AACCGCATCGCAAGAGTTGTTGGAAAAATGAAAATGAGGTATCATAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAACAGCACCTGGT"
                    "AAATTAAGTATTGAAAAAATGC";

                string_to_dna( str_ascii, str_hptr );
                string_to_dna( ref_ascii, ref_hptr );

                aln::EditDistanceAligner<aln::GLOBAL> aligner;

                SingleTest test;
                nvbio::cuda::thrust_copy_vector(test.str_hvec, str_hvec);
                nvbio::cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
                nvbio::cuda::thrust_copy_vector(test.str_dvec, str_hvec);
                nvbio::cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

                test.full<BLOCKDIM,N,M>( "semi-global", &aligner, "1I1M2I1M3I136M" );             
            }*/

            // do a larger speed test of the Gotoh alignment
            if (false && TEST_MASK & (ED | SW | GOTOH | WFA))
            {
                const uint32 N_TASKS = N_THREAD_TASKS;
                const uint32 M = 100;
                const uint32 N = 400;

                const uint32 M_WORDS = (M + 7) >> 3;
                const uint32 N_WORDS = (N + 15) >> 4;

                thrust::host_vector<uint32> str(M_WORDS * N_TASKS);
                thrust::host_vector<uint32> ref(N_WORDS * N_TASKS);

                LCG_random rand;
                fill_packed_stream<4u>(rand, 4u, M * N_TASKS, nvbio::raw_pointer(str));
                fill_packed_stream<2u>(rand, 4u, N * N_TASKS, nvbio::raw_pointer(ref));

                thrust::device_vector<uint32> str_dvec(str);
                thrust::device_vector<uint32> ref_dvec(ref);
                thrust::device_vector<int16> score_dvec(N_TASKS);

                if (TEST_MASK & ED)
                {
                    fprintf(stderr, "  testing Edit Distance scoring speed...\n");
                    fprintf(stderr, "    %15s : ", "global");
                    {
                        batch_score_profile_all<N, M>(
                            make_edit_distance_aligner<aln::GLOBAL>(),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                    fprintf(stderr, "    %15s : ", "semi-global");
                    {
                        batch_score_profile_all<N, M>(
                            make_edit_distance_aligner<aln::SEMI_GLOBAL>(),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                    fprintf(stderr, "    %15s : ", "local");
                    {
                        batch_score_profile_all<N, M>(
                            make_edit_distance_aligner<aln::LOCAL>(),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                }
                if (TEST_MASK & ED)
                {
                    aln::SimpleSmithWatermanScheme scoring;
                    scoring.m_match = 2;
                    scoring.m_mismatch = -1;

                    fprintf(stderr, "  testing Hamming Distance scoring speed...\n");
                    fprintf(stderr, "    %15s : ", "semi-global");
                    {
                        batch_score_profile_all<N, M>(
                            make_hamming_distance_aligner<aln::SEMI_GLOBAL>(scoring),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                    fprintf(stderr, "    %15s : ", "local");
                    {
                        batch_score_profile_all<N, M>(
                            make_hamming_distance_aligner<aln::LOCAL>(scoring),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                }
                if (TEST_MASK & SW)
                {
                    aln::SimpleSmithWatermanScheme scoring;
                    scoring.m_match = 2;
                    scoring.m_mismatch = -1;
                    scoring.m_deletion = -1;
                    scoring.m_insertion = -1;

                    fprintf(stderr, "  testing Smith-Waterman scoring speed...\n");
                    fprintf(stderr, "    %15s : ", "global");
                    {
                        batch_score_profile_all<N, M>(
                            make_smith_waterman_aligner<aln::GLOBAL>(scoring),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                    fprintf(stderr, "    %15s : ", "semi-global");
                    {
                        batch_score_profile_all<N, M>(
                            make_smith_waterman_aligner<aln::SEMI_GLOBAL>(scoring),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                    fprintf(stderr, "    %15s : ", "local");
                    {
                        batch_score_profile_all<N, M>(
                            make_smith_waterman_aligner<aln::LOCAL>(scoring),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                }
                if (TEST_MASK & GOTOH)
                {
                    aln::SimpleGotohScheme scoring;
                    scoring.m_match = 2;
                    scoring.m_mismatch = -1;
                    scoring.m_gap_open = -1;
                    scoring.m_gap_ext = -1;

                    fprintf(stderr, "  testing Gotoh scoring speed...\n");
                    fprintf(stderr, "    %15s : ", "global");
                    {
                        batch_score_profile_all<N, M>(
                            make_gotoh_aligner<aln::GLOBAL>(scoring),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                    fprintf(stderr, "    %15s : ", "semi-global");
                    {
                        batch_score_profile_all<N, M>(
                            make_gotoh_aligner<aln::SEMI_GLOBAL>(scoring),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                    fprintf(stderr, "    %15s : ", "local");
                    {
                        batch_score_profile_all<N, M>(
                            make_gotoh_aligner<aln::LOCAL>(scoring),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                }
                if (false && TEST_MASK & WFA)
                {
                    aln::SimpleWfahScheme scoring;
                    scoring.m_match = 0;
                    scoring.m_mismatch = 2;
                    scoring.m_gap_open = 4;
                    scoring.m_gap_ext = 1;

                    fprintf(stderr, "  testing Wfa scoring speed...\n");
                    fprintf(stderr, "    %15s : ", "global");
                    {
                        batch_score_profile_all<N, M>(
                            make_wfah_aligner<aln::GLOBAL>(scoring),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec);
                    }
                    /*fprintf(stderr,"    %15s : ", "semi-global");
                    {
                        batch_score_profile_all<N,M>(
                            make_wfah_aligner<aln::SEMI_GLOBAL>( scoring ),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }
                    fprintf(stderr,"    %15s : ", "local");
                    {
                        batch_score_profile_all<N,M>(
                            make_wfah_aligner<aln::LOCAL>( scoring ),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }*/
                }
            }
            // do a larger speed test of the banded SW alignment
            if (false && TEST_MASK & (ED_BANDED | SW_BANDED | GOTOH_BANDED | WFA_BANDED))
            {
                const uint32 BAND_LEN = 31u;
                const uint32 N_TASKS  = N_THREAD_TASKS;
                const uint32 M = 150;
                const uint32 N = M+BAND_LEN;

                const uint32 M_WORDS = (M + 7)  >> 3;
                const uint32 N_WORDS = (N + 15) >> 4;

                thrust::host_vector<uint32> str( M_WORDS * N_TASKS );
                thrust::host_vector<uint32> ref( N_WORDS * N_TASKS );

                LCG_random rand;
                fill_packed_stream<4u>( rand, 4u, M * N_TASKS, nvbio::raw_pointer( str ) );
                fill_packed_stream<2u>( rand, 4u, N * N_TASKS, nvbio::raw_pointer( ref ) );

                thrust::device_vector<uint32> str_dvec( str );
                thrust::device_vector<uint32> ref_dvec( ref );
                thrust::device_vector<int16>  score_dvec( N_TASKS );

                if (TEST_MASK & ED_BANDED)
                {
                    fprintf(stderr,"  testing banded Edit Distance scoring speed...\n");
                    fprintf(stderr,"    %15s : ", "global");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_edit_distance_aligner<aln::GLOBAL>(),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }
                    fprintf(stderr,"    %15s : ", "semi-global");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_edit_distance_aligner<aln::SEMI_GLOBAL>(),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }
                    fprintf(stderr,"    %15s : ", "local");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_edit_distance_aligner<aln::LOCAL>(),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }
                }
                if (TEST_MASK & SW_BANDED)
                {
                    fprintf(stderr,"  testing banded Smith-Waterman scoring speed...\n");
                    fprintf(stderr,"    %15s : ", "global");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_smith_waterman_aligner<aln::GLOBAL>( aln::SimpleSmithWatermanScheme(2,-1,-1,-1) ),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }
                    fprintf(stderr,"    %15s : ", "semi-global");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_smith_waterman_aligner<aln::SEMI_GLOBAL>( aln::SimpleSmithWatermanScheme(2,-1,-1,-1) ),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }
                    fprintf(stderr,"    %15s : ", "local");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_smith_waterman_aligner<aln::LOCAL>( aln::SimpleSmithWatermanScheme(2,-1,-1,-1) ),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }
                }
                if (TEST_MASK & GOTOH_BANDED)
                {
                    fprintf(stderr,"  testing banded Gotoh scoring speed...\n");
                    fprintf(stderr,"    %15s : ", "global");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_gotoh_aligner<aln::GLOBAL>( aln::SimpleGotohScheme(2,-1,-1,-1) ),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }
                    fprintf(stderr,"    %15s : ", "semi-global");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_gotoh_aligner<aln::SEMI_GLOBAL>( aln::SimpleGotohScheme(2,-1,-1,-1) ),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }
                    fprintf(stderr,"    %15s : ", "local");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_gotoh_aligner<aln::LOCAL>( aln::SimpleGotohScheme(2,-1,-1,-1) ),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }
                }
                if (TEST_MASK & WFA_BANDED)
                {
                    fprintf(stderr,"  testing banded Wfa scoring speed...\n");
                    /*fprintf(stderr,"    %15s : ", "global");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_wfah_aligner<aln::GLOBAL>( aln::SimpleWfahScheme(0,2,4,1) ),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }*/
                    fprintf(stderr,"    %15s : ", "semi-global");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_wfah_aligner<aln::SEMI_GLOBAL>( aln::SimpleWfahScheme(0,2,4,1) ),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }
                    /*fprintf(stderr,"    %15s : ", "local");
                    {
                        batch_banded_score_profile_all<BAND_LEN,N,M>(
                            make_wfah_aligner<aln::LOCAL>( aln::SimpleWfahScheme(0,2,4,1) ),
                            n_tests,
                            N_TASKS,
                            str_dvec,
                            ref_dvec,
                            score_dvec );
                    }*/
                }

            }
            fprintf(stderr, "testing alignment... done\n");
        }

    } // namespace sw
} // namespace nvbio
